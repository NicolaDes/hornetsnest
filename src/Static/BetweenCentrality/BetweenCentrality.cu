#include "hip/hip_runtime.h"
#include "Static/BetweenCentrality/BetweenCentrality.cuh"
#include "Core/Auxilary/DuplicateRemoving.cuh"
#include <Graph/GraphStd.hpp>
#include <Graph/BC.hpp>

namespace hornets_nest
{

const dist_t INF = std::numeric_limits<dist_t>::max();

//------------------------------------------------------------------------------
///////////////
// OPERATORS //
///////////////

// O(max(queue.size)) -> worst case is when all edges go in one vertice
struct SSP
{
    dist_t *d_distances;
    dist_t current_level;
    float *sigma;
    TwoLevelQueue<vid_t> queue;

    OPERATOR(Vertex &vertex, Edge &edge)
    {
        auto dst = edge.dst_id();
        auto src = edge.src_id();

        if (atomicCAS(&d_distances[dst], INF,current_level+1) == INF)
        {   
            queue.insert(dst);
        }

        if (d_distances[dst] == d_distances[src] + 1)
        {
            atomicAdd(&sigma[dst], sigma[src]);
        }
    }
};

// O(max(queue.size)) -> the same of SSP
struct ACCUMULATE
{
    dist_t *d_distances;
    float *sigma;
    float *delta;
    OPERATOR(Vertex &vertex, Edge &edge)
    {
	        auto dst = edge.dst_id();
                auto src = edge.src_id();
                float add=(sigma[src] / sigma[dst]) * (1 + delta[dst]);
                if (d_distances[dst] == d_distances[src] + 1)
                {
                  atomicAdd(&delta[src], add);
                }

                
    }
};

// O(1) -> it's an array and each thread works on a single element
struct LOAD_S
{
    vid_t* S;
    int offset_length;
    OPERATOR(Vertex &vertex)
    {
	int tid = blockIdx.x*blockDim.x+threadIdx.x;	
        S[tid + offset_length] = vertex.id();
    }
};

// O(1) -> the same of LOAD_S, but queue.insert contains atomic operations?
struct LOAD_QUEUE
{
    int left_limit, right_limit;
    vid_t* S;
    TwoLevelQueue<vid_t> queue;
    OPERATOR(Vertex &vertex)
    {
	int i=blockIdx.x*blockDim.x+threadIdx.x;
        if (i <= right_limit && i >= left_limit)
                {
                    queue.insert(S[i]);
                }
    }
};

BetweenCentrality::BetweenCentrality(HornetGraph &hornet) : StaticAlgorithm(hornet),
                                                            queue(hornet),
                                                            load_balancing(hornet)
{
    gpu::allocate(d_distances, hornet.nV());
    gpu::allocate(sigma, hornet.nV());
    gpu::allocate(S, hornet.nV());
    gpu::allocate(delta, hornet.nV());
    gpu::allocate(bc, hornet.nV());
    reset();
}

void BetweenCentrality::run()	
{
    int offset_length = 0;
    vid_t tid;
  
	int num_nodes=hornet.nV();

    for (tid = 0; tid < hornet.nV(); tid++)	
    {

    int *ends = new int[hornet.nV() + 1];
        //INIT CONFIG
        ends[0] = 0;
        ends[1] = 1;
        int ends_l = 2;

        //CLEAR AND RESET
        this->reset();
        this->set_parameters(tid);


        current_level = 0;

        offset_length=0;
        //BFS
        while (queue.size() > 0)		
        { 
            forAllEdges(hornet, queue, SSP{d_distances, current_level, sigma, queue}, load_balancing);

            forAllVertices(hornet, queue, LOAD_S{S, offset_length});
            
            current_level++;
            offset_length += queue.size();

            queue.swap();
            ends[ends_l] = ends[ends_l - 1] + queue.size();
            
            ends_l++;
        }

        //ACCUMULATION STEP
        while (current_level > 0)		
        {   
            int left_limit = ends[current_level];
            int right_limit = ends[current_level + 1] - 1;
            forAllVertices(hornet, LOAD_QUEUE{left_limit, right_limit, S, queue});	
            queue.swap();          
            if(queue.size()>0) forAllEdges(hornet, queue, ACCUMULATE{d_distances, sigma, delta}, load_balancing); 
            current_level--;
        }

        auto delta_ = delta;
        auto bc_d=bc;
        forAllnumV(hornet, [=] __device__(int i) {bc_d[i]=bc_d[i]+delta_[i];});	
	delete[] ends;

    }
 }

BetweenCentrality::~BetweenCentrality()
{
    gpu::free(d_distances);
    gpu::free(delta);
    gpu::free(sigma);
    gpu::free(S);
    gpu::free(bc);
}

void BetweenCentrality::reset()
{
    current_level = 1;
    queue.clear();

    auto distances = d_distances;
    forAllnumV(hornet, [=] __device__(int i) { distances[i] = INF; });
    auto delta_ = delta;
    forAllnumV(hornet, [=] __device__(int i) { delta_[i] = 0; });
}

void BetweenCentrality::set_parameters(vid_t source)
{
    bfs_source = source;
    queue.insert(bfs_source);                  // insert bfs source in the frontier
    gpu::memsetZero(d_distances + bfs_source); //reset source distance
    auto distances = d_distances;
    forAllnumV(hornet, [=] __device__(int i) { distances[i] = INF; distances[source] = 0;});
    auto sigma_ = sigma;
    forAllnumV(hornet, [=] __device__(int i) {sigma_[i]=0;sigma_[source]=1; });
}

void BetweenCentrality::release()
{
    gpu::free(d_distances);
    gpu::free(sigma);
    gpu::free(S);
    gpu::free(delta);
    gpu::free(bc);
    d_distances = nullptr;
    S = nullptr;
    sigma = nullptr;
    delta = nullptr;
    bc=nullptr;
}
template<typename HostIterator, typename DeviceIterator>
bool equal(HostIterator host_start, HostIterator host_end,
           DeviceIterator device_start) noexcept {
    using R = typename std::iterator_traits<DeviceIterator>::value_type;
    auto size = std::distance(host_start, host_end);
    R* array = new R[size];
    cuMemcpyToHost(&(*device_start), size, array);

    bool flag;// = std::equal(host_start, host_end, array);
    flag=true;
    float epsilon=1;
    float maxErr=0;
    int index=0;
    if (true/*!flag*/) {
        for (int i = 0; i < size; i++) {
		float tmp=abs(host_start[i]-array[i]);
		if(maxErr<tmp){
			maxErr=tmp;
			index=i;
		}
            if (host_start[i]-array[i]>epsilon||host_start[i]-array[i]<-epsilon) {
                std::cout << std::setprecision(13) << host_start[i] << "  " << std::setprecision(13) << array[i] << "  at "
                          << i <<" with err value: "<<abs(host_start[i]-array[i])<< std::endl;
                flag=false;
            }
        }
    }
//    printf("MAX error precision: %.20f\t, on a total value of: %.10f\n", maxErr, host_start[index]);
    delete[] array;
    return flag;
}


bool BetweenCentrality::validate(timer::Timer<timer::DEVICE> TM)
{
    using namespace graph;
    GraphStd<vid_t, eoff_t> graph(hornet.csr_offsets(), hornet.nV(),
                                  hornet.csr_edges(), hornet.nE());
    BC<vid_t, eoff_t> betcen(graph);
    timer::Timer<timer::HOST> TM_H;
    TM_H.start();
  
    betcen.run(bfs_source);

    
    TM_H.stop();
    TM_H.print("Sequential: ");  
    printf("SpeedUP: %f\n",TM_H.duration()/TM.duration());
    auto sequentail_bc = betcen.result();
    return equal(sequentail_bc, sequentail_bc + graph.nV(), bc);
}
bool BetweenCentrality::validate()
{
    using namespace graph;
    GraphStd<vid_t, eoff_t> graph(hornet.csr_offsets(), hornet.nV(),
                                  hornet.csr_edges(), hornet.nE());
    BC<vid_t, eoff_t> betcen(graph);
    betcen.run(bfs_source);

    auto sequentail_bc = betcen.result();
    return gpu::equal(sequentail_bc, sequentail_bc + graph.nV(), bc);
}
} // namespace hornets_nest
