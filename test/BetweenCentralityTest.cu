/**
 * @brief Between centrality
 * @file
 */
#include "Static/BetweenCentrality/BetweenCentrality.cuh"
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

int main(int argc, char* argv[]) {
    using namespace timer;
    using namespace graph;
    using namespace hornets_nest;
    using namespace structure_prop;

    graph::GraphStd<vid_t, eoff_t> graph(ENABLE_INGOING);
    CommandLineParam cmd(graph, argc, argv);
    //graph.print();

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);

    BetweenCentrality betcen(hornet_graph);


    Timer<DEVICE> TM;
       hipProfilerStart();
       TM.start();
   //    betcen.set_parameters(graph.max_out_degree_id());
       betcen.run();
   
       TM.stop();
       hipProfilerStop();
       TM.print("Parallel: ");



	//graph.print();
    auto is_correct = betcen.validate(TM);

  std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");


return 0;
}
